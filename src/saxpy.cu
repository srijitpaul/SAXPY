#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>


__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i] = a*x[i] + y[i];
	}
}

double min(double* array, int size){
    // returns the minimum value of array
    double val = array[0];
    for (int i = 1; i < size; ++i){
        val = val <= array[i] ? val : array[i];
    }
    return val;
}

double max(double* array, int size){
    // returns the maximum value of array
    double val = array[0];
    for (int i = 1; i < size; ++i){
        val = val >= array[i] ? val : array[i];
    }
    return val;
}

double mean(double* array, int size) {
    double sum=0;
    for(int i=0; i<size; i++)
        sum+=array[i];
    return((double)sum/size);
}

int main(int argc, char * argv[])
{
	unsigned long int arrlength= atoi(argv[1]);
	unsigned long int N = 1<<arrlength;
	unsigned long int nruns = atoi(argv[2]);
	size_t size = N*sizeof(float);
	printf("N = %d\n",N);
	double ntime[nruns],nbandwidth[nruns];
	for(unsigned long int count = 0; count < nruns; count ++){
		// Create CUDA events for timing purposes
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		float *x, *y;		// Host vectors
		float *d_x, *d_y;	// Device vectors

		// Allocate host memory
		x = (float *)malloc(size);
		y = (float *)malloc(size);

		// Allocate device memory
		hipMalloc(&d_x, size);
		hipMalloc(&d_y, size);

		for (unsigned long int i = 0; i < N; i++){
			x[i] = 1.0f;
			y[i] = 2.0f;
		}

		hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

		// Perform SAXPY on 1M elements
		hipEventRecord(start);
		saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
		hipEventRecord(stop);

		hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		float maxError = 0.0f;
		for (unsigned long int i = 0; i < N; i++){
			maxError = max(maxError, abs(y[i]-4.0f));
		}
		printf("Max error: %f\n", maxError);
		ntime[count] = milliseconds;
		nbandwidth[count] = N*4*3/milliseconds/1e6;
	        hipFree(d_x);
		hipFree(d_y);
		free(x);
		free(y);	
	}
	printf("Average time of execution: %f\n", mean(ntime,nruns));
	printf("Maximum time of execution: %f\n", max(ntime,nruns));
	printf("Mininum time of execution: %f\n", min(ntime,nruns));
	printf("Average Bandwidth: %f\n", mean(nbandwidth,nruns));
	printf("Maximum Bandwidth: %f\n", max(nbandwidth,nruns));
	printf("Mininum Bandwidth: %f\n", min(nbandwidth,nruns));

}
