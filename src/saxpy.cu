#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>


__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i] = a*x[i] + y[i];
	}
}

int main()
{
	int N = 1<<27;
	int size = N*sizeof(float);
	printf("N = %d\n",N);

	// Create CUDA events for timing purposes
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *x, *y;		// Host vectors
	float *d_x, *d_y;	// Device vectors

	// Allocate host memory
	x = (float *)malloc(size);
	y = (float *)malloc(size);

	// Allocate device memory
	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);

	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	// Perform SAXPY on 1M elements
	hipEventRecord(start);
	saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
	hipEventRecord(stop);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++) 
	{
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	printf("Succesfully performed SAXPY on %d elements in %f milliseconds.\n", N, milliseconds);
	printf("Effective Bandwidth (GB/s): %f\n", N*4*3/milliseconds/1e6);
}
