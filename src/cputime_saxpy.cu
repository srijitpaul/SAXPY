#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
using namespace std;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a*x[i] + y[i];
    }
}

int main(int argc, char * argv[])
{
    int arrlength= atoi(argv[1]);
    int N = 1<<arrlength;
    int nruns = atoi(argv[2]);
    int size = N*sizeof(float);
 
    //create stop timers
    double wall_timestop_1, wall_timestop_2, wall_timestop_3;

    float *x, *y;		// Host vectors
    float *d_x, *d_y;	// Device vectors

    // Allocate host memory
    x = (float *)malloc(size);
    y = (float *)malloc(size);


  
    // Allocate device memory
   

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    double wall_timestart_1 = get_wall_time();

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    for(int count = 0; count < nruns; count ++){

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
    }
        
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    wall_timestop_1 = get_wall_time();

    double seconds_1;


    seconds_1 = wall_timestop_1 - wall_timestart_1;

    double wall_timestart_2 = get_wall_time();

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    for(int count = 0; count < nruns; count ++){

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
    }
        
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    wall_timestop_2 = get_wall_time();

    double seconds_2;


    seconds_2 = wall_timestop_2 - wall_timestart_2;

    double wall_timestart_3 = get_wall_time();

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    for(int count = 0; count < nruns; count ++){

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
    }
        
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    wall_timestop_3 = get_wall_time();

    double seconds_3;


    seconds_3 = wall_timestop_3 - wall_timestart_3;

    double mean_seconds = (seconds_1 + seconds_2 + seconds_3)/3;

    cout<<nruns<<"\t\t"<<mean_seconds<<"\t\t"<<sqrt((pow((seconds_1 - mean_seconds),2) + pow((seconds_2 - mean_seconds),2) + pow((seconds_3 - mean_seconds),2))/3)<<endl;

}
