#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
using namespace std;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a*x[i] + y[i];
    }
}

int main(int argc, char * argv[])
{
    int arrlength= atoi(argv[1]);
    int N = 1<<arrlength;
    int nruns = atoi(argv[2]);
    int size = N*sizeof(float);
    printf("N = %d\n",N);
    char *output_file;
    output_file = new char[1024];
    output_file = argv[3];
    ofstream outfile(output_file,ios::out);
    //create stop timers
    double wall_timestop[nruns];

    float *x, *y;		// Host vectors
    float *d_x, *d_y;	// Device vectors

    // Allocate host memory
    x = (float *)malloc(size);
    y = (float *)malloc(size);


    double wall_timestart = get_wall_time();
    // Allocate device memory
    for(int count = 0; count < nruns; count ++){

    	hipMalloc(&d_x, size);
    	hipMalloc(&d_y, size);

    	for (int i = 0; i < N; i++){
        	x[i] = 1.0f;
        	y[i] = 2.0f;
    	}


   
        hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
        hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
        wall_timestop[count] = get_wall_time();
    }
    double seconds[nruns];

    for(int i = 0; i < nruns; i++){
        seconds[i] = wall_timestop[i] - wall_timestart;
        outfile<<i+1<<"\t\t"<<seconds[i]<<endl;
    }

}
