#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
using namespace std;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a*x[i] + y[i];
    }
}


double min(double* array, int size){
    // returns the minimum value of array
    double val = array[0];
    for (int i = 1; i < size; ++i){
        val = val <= array[i] ? val : array[i];
    }
    return val;
}

int main(int argc, char * argv[])
{
    unsigned long int arrlength= atoi(argv[1]);
    unsigned long int N = 1<<arrlength;
    unsigned long int nruns = atoi(argv[2]);
    unsigned long int neval = atoi(argv[3]);
    size_t size = N*sizeof(float);
    double seconds[neval];
 

    float *x, *y;		// Host vectors
    float *d_x, *d_y;	// Device vectors

    // Allocate host memory
    x = (float *)malloc(size);
    y = (float *)malloc(size);


  
    // Allocate device memory
   

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    for (unsigned long int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
        
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);


    for(unsigned long int run = 0; run < neval; run++)
    {

    	// Allocate device memory
   

    	hipMalloc(&d_x, size);
    	hipMalloc(&d_y, size);

	

    	double wall_timestart = get_wall_time();

    	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    	for(unsigned long int count = 0; count < nruns; count ++){

        	// Perform SAXPY on 1M elements
        	saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
    	}
        
    	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    	double wall_timestop = get_wall_time();

    	hipFree(d_x);
    	hipFree(d_y);


    	seconds[run] = wall_timestop - wall_timestart;


    }

    free(x);
    free(y);
    cout<<nruns<<"\t\t"<<min(seconds,neval)<<endl;
 
    

}
