#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <string>
#include <cupti_profiler.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i] = a*x[i] + y[i];
	}
}

double min(double* array, int size){
	// returns the minimum value of array
	double val = array[0];
	for (int i = 1; i < size; ++i){
		val = val <= array[i] ? val : array[i];
	}
	return val;
}

double max(double* array, int size){
	// returns the maximum value of array
	double val = array[0];
	for (int i = 1; i < size; ++i){
		val = val >= array[i] ? val : array[i];
	}
	return val;
}

double mean(double* array, int size) {
	double sum=0;
	for(int i=0; i<size; i++)
		sum+=array[i];
	return((double)sum/size);
}

int main(int argc, char * argv[])
{
	using namespace std;

	vector<string> event_names {
		"active_warps",
			"gst_inst_32bit",
			"gld_inst_32bit",
			"warps_launched",
	//		"threads_launched",
	//		"gst_32b",
	//		"gld_32b",
	//		"sm_cta_launched",
			"branch",
	//		"divergent_branch",
	//		"NAME",
			"active_cycles"
	};
	vector<string> metric_names {
	//	"flop_count_dp",
			"flop_count_sp",
			"inst_executed",
			"dram_read_transactions",
			"dram_write_transactions",
	//		"dram_read_throughput",
       	//		"dram_write_throughput",
	//		"l2_read_throughput",
	//		"l2_write_throughput",
			"l2_read_transactions",
			"l2_write_transactions",
			"l2_tex_read_transactions",
			"l2_tex_write_transactions"			
				//"stall_memory_throttle"
	};
	cupti_profiler::profiler profiler(event_names, metric_names);

	// Get #passes required to compute all metrics and events
	const int passes = profiler.get_passes();
	printf("Passes: %d\n", passes);

	unsigned long int arrlength= atoi(argv[1]);
	unsigned long int N = 1<<arrlength;
	unsigned long int nruns = atoi(argv[2]);
	size_t size = N*sizeof(float);
	printf("N = %d\n",N);
	float *x, *y;		// Host vectors
	float *d_x, *d_y;	// Device vectors

	// Allocate host memory
	x = (float *)malloc(size);
	y = (float *)malloc(size);

	// Allocate device memory
	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);

	for (unsigned long int i = 0; i < N; i++){
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	// Perform SAXPY on 1M elements

	saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);


	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);	

	double ntime[nruns],nbandwidth[nruns];
	profiler.start();
	for(unsigned long int count = 0; count < nruns; count ++){
		// Create CUDA events for timing purposes
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);




		// Allocate host memory
		x = (float *)malloc(size);
		y = (float *)malloc(size);

		// Allocate device memory
		hipMalloc(&d_x, size);
		hipMalloc(&d_y, size);

		for (unsigned long int i = 0; i < N; i++){
			x[i] = 1.0f;
			y[i] = 2.0f;
		}

		hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

		// Perform SAXPY on 1M elements
		hipEventRecord(start);
		saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
		hipEventRecord(stop);
		hipDeviceSynchronize();
		hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

		hipEventSynchronize(stop);

				float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		float maxError = 0.0f;
		for (unsigned long int i = 0; i < N; i++){
			maxError = max(maxError, abs(y[i]-4.0f));
		}
		//	printf("Max error: %f\n", maxError);
				ntime[count] = milliseconds;
				nbandwidth[count] = N*4*3/milliseconds/1e6;
		hipFree(d_x);
		hipFree(d_y);
		free(x);
		free(y);	
	}
	profiler.stop();
	printf("Event Trace\n");
	profiler.print_event_values(std::cout);
	printf("Metric Trace\n");
	profiler.print_metric_values(std::cout);

	auto names = profiler.get_kernel_names();
	for(auto name: names) {
		printf("%s\n", name.c_str());
	}
	//printf("Average time of execution: %f\n", mean(ntime,nruns));
	//printf("Maximum time of execution: %f\n", max(ntime,nruns));
	printf("Mininum time of execution: %f\n", min(ntime,nruns));
	//printf("Average Bandwidth: %f\n", mean(nbandwidth,nruns));
	printf("Maximum Bandwidth: %f\n", max(nbandwidth,nruns));
	//printf("Mininum Bandwidth: %f\n", min(nbandwidth,nruns));

}
